#include "hip/hip_runtime.h"
#include <sys/time.h>
#include <stdio.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

#include "basic_structure.h"

__global__ void
cuda_blur_filter_kernel(int *p, int * res, int size, int threshold, int width, int height, int* end){
    int i, j, k;
    j = threadIdx.y + blockIdx.y * blockDim.y;
    k = threadIdx.x + blockIdx.x * blockDim.x;
    i = CONV(j,k,width);
    int end1 = height/10 - size;
    int end2 = height*0.9 + size;

    if (j >= 0 && j < height-1 && k >= 0 && k < width-1)
    {
        res[i] = p[i];
    }
    __syncthreads();
    /* Apply blur on top/bottom part of image (10% & 90%) */
    if (k >= size && k < width-size){
        if ((j >= size && j < end1) || (j >= end2 && j < height-size))
        {
            int stencil_j, stencil_k ;
            int t_r = 0 ;
            for ( stencil_j = -size ; stencil_j <= size ; stencil_j++ )
            {
                for ( stencil_k = -size ; stencil_k <= size ; stencil_k++ )
                {
                    t_r += p[CONV(j+stencil_j,k+stencil_k,width)] ;
                }
            }
            res[i] = t_r / ( (2*size+1)*(2*size+1) ) ;
            /* Test the end condition on the variables that we have changed */
            float diff_r;
            diff_r = (res[i] - p[i]);
            if ( diff_r > threshold || -diff_r > threshold) {
                *end = 0;
            }
        }
        /* Copy the middle part of the image */
        else if (j >= end1 && j < end2){
            res[i] = p[i];
        }
    }
    __syncthreads();
    /* If the difference is large enough, we are going to reblur the image */
    if (j >= 1 && j < height - 1 && k >= 1 && k < width - 1)
    {
        p[i] = res[i] ;
    }
    __syncthreads();
}
__global__
void cuda_sobel_filter_kernel(int* p, int* res, int width, int height){
    int i, j, k;
    j = threadIdx.y + blockIdx.y * blockDim.y;
    k = threadIdx.x + blockIdx.x * blockDim.x;
    i = CONV(j,k,width);
    if (j >= 1 && j < height - 1 && k >= 1 && k < width-1)
    {
        int pixel_blue_no, pixel_blue_n, pixel_blue_ne;
        int pixel_blue_so, pixel_blue_s, pixel_blue_se;
        int pixel_blue_o , pixel_blue_e ;
        float deltaX_blue ;
        float deltaY_blue ;
        float val_blue;
        pixel_blue_no = p[CONV(j-1,k-1,width)] ;
        pixel_blue_n  = p[CONV(j-1,k  ,width)] ;
        pixel_blue_ne = p[CONV(j-1,k+1,width)] ;
        pixel_blue_so = p[CONV(j+1,k-1,width)] ;
        pixel_blue_s  = p[CONV(j+1,k  ,width)] ;
        pixel_blue_se = p[CONV(j+1,k+1,width)] ;
        pixel_blue_o  = p[CONV(j  ,k-1,width)] ;
        pixel_blue_e  = p[CONV(j  ,k+1,width)] ;

        deltaX_blue = -pixel_blue_no + pixel_blue_ne - 2*pixel_blue_o + 2*pixel_blue_e - pixel_blue_so + pixel_blue_se;
        deltaY_blue = pixel_blue_se + 2*pixel_blue_s + pixel_blue_so - pixel_blue_ne - 2*pixel_blue_n - pixel_blue_no;
        val_blue = sqrt(deltaX_blue * deltaX_blue + deltaY_blue * deltaY_blue)/4;
        if ( val_blue > 50 ) res[i] = 255 ;
        else res[i] = 0 ;
    }
    else if (j < height && k < width)
    {
        res[i] = p[i];
    }
    __syncthreads();
}

extern "C"
{
    int cuda_blur_in_while(int* p, int size, int threshold, int width, int height)
    {
        hipSetDevice(0);
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, 0);

        int total_size = width * height * sizeof(int);

        dim3 dimBlock(
                min(32, width), 
                min(32, height)
                );
        dim3 dimGrid(
                width / dimBlock.x + 1,
                height / dimBlock.y + 1
                );
#if CUDA_DEBUG
        printf("\nSize dimBlock : %d x %d \n", dimBlock.x, dimBlock.y);
        printf("Size dimGrid : %d x %d \n", dimGrid.x, dimGrid.y);
        printf("Threads needed : %d, Threads had : %d \n", width * height, dimBlock.x*dimBlock.y*dimGrid.x * dimGrid.y);
#endif
        /* Define device variables */
        int * d_p;
        int * d_res;
        int * d_end;

        /* Allocation of memory */
        checkCudaErrors(hipMalloc( &d_p, total_size));
        checkCudaErrors(hipMalloc( &d_res, total_size));
        checkCudaErrors(hipMalloc( &d_end, sizeof(int)));

        /* Copy array from CPU to device */
        checkCudaErrors(hipMemcpy(d_p, p, total_size, hipMemcpyHostToDevice));

        /* execute the kernel */
        int end;
        end = 1;
        hipMemcpy(d_end, &end, sizeof(int), hipMemcpyHostToDevice);
        cuda_blur_filter_kernel<<<dimGrid, dimBlock>>>(d_p, d_res, size, threshold, width, height, d_end);
        hipMemcpy(&end, d_end, sizeof(int), hipMemcpyDeviceToHost);

        /* return the result from device to CPU */
        checkCudaErrors(hipMemcpy(p, d_res, total_size, hipMemcpyDeviceToHost));
        checkCudaErrors(hipFree(d_p));
        checkCudaErrors(hipFree(d_res));
        checkCudaErrors(hipFree(d_end));
        return end;
    }

    void cuda_blur_filter_per_image(int* p, int size, int threshold, int width, int height)
    {
        hipSetDevice(0);
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, 0);

        int total_size = width * height * sizeof(int);

        dim3 dimBlock(
                min(32, width), 
                min(32, height)
                );
        dim3 dimGrid(
                width / dimBlock.x + 1,
                height / dimBlock.y + 1
                );
#if CUDA_DEBUG
        printf("\nSize dimBlock : %d x %d \n", dimBlock.x, dimBlock.y);
        printf("Size dimGrid : %d x %d \n", dimGrid.x, dimGrid.y);
        printf("Threads needed : %d, Threads had : %d \n", width * height, dimBlock.x*dimBlock.y*dimGrid.x * dimGrid.y);
#endif
        /* Define device variables */
        int * d_p;
        int * d_res;
        int * d_end;

        /* Allocation of memory */
        checkCudaErrors(hipMalloc( &d_p, total_size));
        checkCudaErrors(hipMalloc( &d_res, total_size));
        checkCudaErrors(hipMalloc( &d_end, sizeof(int)));

        /* Copy array from CPU to device */
        checkCudaErrors(hipMemcpy(d_p, p, total_size, hipMemcpyHostToDevice));

        /* execute the kernel */
        int num_iter = 0;
        int end;
        do{
            end = 1;
            num_iter++;
            hipMemcpy(d_end, &end, sizeof(int), hipMemcpyHostToDevice);
            cuda_blur_filter_kernel<<<dimGrid, dimBlock>>>(d_p, d_res, size, threshold, width, height, d_end);
            hipMemcpy(&end, d_end, sizeof(int), hipMemcpyDeviceToHost);
        }while (threshold > 0 && !end);
#if CUDA_DEBUG
        printf("\nBlur filtering...Done! %d \n", num_iter);
#endif

        /* return the result from device to CPU */
        checkCudaErrors(hipMemcpy(p, d_res, total_size, hipMemcpyDeviceToHost));
        checkCudaErrors(hipFree(d_p));
        checkCudaErrors(hipFree(d_res));
        checkCudaErrors(hipFree(d_end));
    }

    void cuda_sobel_filter_per_image(int* p, int width, int height){
        hipSetDevice(0);
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, 0);

        int total_size = width * height * sizeof(int);
        //printf("\n %d \n", total_size);

        dim3 dimBlock(
                min(32, width), 
                min(32, height)
                );
        dim3 dimGrid(
                width / dimBlock.x + 1,
                height / dimBlock.y + 1
                );
#if CUDA_DEBUG
        printf("\nSize dimBlock : %d x %d \n", dimBlock.x, dimBlock.y);
        printf("Size dimGrid : %d x %d \n", dimGrid.x, dimGrid.y);
        printf("Threads needed : %d, Threads had : %d \n", width * height, dimBlock.x*dimBlock.y*dimGrid.x * dimGrid.y);
#endif
        /* Define device variables */
        int * d_p;
        int * d_res;

        /* Allocation of memory */
        checkCudaErrors(hipMalloc( &d_p, total_size));
        checkCudaErrors(hipMalloc( &d_res, total_size));

        /* Copy array from CPU to device */
        checkCudaErrors(hipMemcpy(d_p, p, total_size, hipMemcpyHostToDevice));

        /* execute the kernel */
        cuda_sobel_filter_kernel<<<dimGrid, dimBlock>>>(d_p, d_res, width, height);

        /* return the result from device to CPU */
        checkCudaErrors(hipMemcpy(p, d_res, total_size, hipMemcpyDeviceToHost));
        checkCudaErrors(hipFree(d_p));
        checkCudaErrors(hipFree(d_res));
    }

    void cuda_filter_per_image(int* p, int size, int threshold, int width, int height){
        hipSetDevice(0);
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, 0);
        hipEvent_t start, stop;

        int total_size = width * height * sizeof(int);
        //printf("\n %d \n", total_size);

        dim3 dimBlock(
                min(32, width), 
                min(32, height)
                );
        dim3 dimGrid(
                width / dimBlock.x + 1,
                height / dimBlock.y + 1
                );
#if CUDA_DEBUG
        printf("\nSize dimBlock : %d x %d \n", dimBlock.x, dimBlock.y);
        printf("Size dimGrid : %d x %d \n", dimGrid.x, dimGrid.y);
        printf("Threads needed : %d, Threads had : %d \n", width * height, dimBlock.x*dimBlock.y*dimGrid.x * dimGrid.y);
#endif
        /* Define device variables */
        int * d_p;
        int * d_res;
        int * d_end;

        /* Allocation of memory */
        checkCudaErrors(hipEventCreate(&start));
        checkCudaErrors(hipMalloc( &d_p, total_size));
        checkCudaErrors(hipMalloc( &d_res, total_size));
        checkCudaErrors(hipMalloc( &d_end, sizeof(int)));
        checkCudaErrors(hipEventCreate(&stop));

        /* Copy array from CPU to device */
        checkCudaErrors(hipMemcpy(d_p, p, total_size, hipMemcpyHostToDevice));

        /* execute the kernel */
        int num_iter = 0;
        int end;
        do{
            end = 1;
            num_iter++;
            hipMemcpy(d_end, &end, sizeof(int), hipMemcpyHostToDevice);
            cuda_blur_filter_kernel<<<dimGrid, dimBlock>>>(d_p, d_res, size, threshold, width, height, d_end);
            hipMemcpy(&end, d_end, sizeof(int), hipMemcpyDeviceToHost);
        }while (threshold > 0 && !end);
#if CUDA_DEBUG
        printf("\nBlur filtering...Done! %d \n", num_iter);
#endif

        cuda_sobel_filter_kernel<<<dimGrid, dimBlock>>>(d_p, d_res, width, height);

        /* return the result from device to CPU */
        checkCudaErrors(hipMemcpy(p, d_res, total_size, hipMemcpyDeviceToHost));
        checkCudaErrors(hipFree(d_p));
        checkCudaErrors(hipFree(d_res));
        checkCudaErrors(hipFree(d_end));
    }

    void cuda_filter( animated_gif * image){
#if SOBELF_DEBUG
        fprintf(stderr, "\nUsing CUDA functions\n");
#endif

#if time_eval
        printf("%s ", "CUDA");
        struct timeval t1, t2;
        double duration;
        /* FILTER Timer start */
        gettimeofday(&t1, NULL);
#endif

    // Apply cuda filter
        int i, width, height;
        int ** p;
        p = image->p;
        for(i=0; i<image->n_images; i++){
            width = image->width[i];
            height = image->height[i];
            //cuda_filter_per_image(p[i], 5, 20, width, height);

#if time_eval_filters
    struct timeval t3, t4;
    double duration2;
    gettimeofday(&t3, NULL);
#endif

            cuda_blur_filter_per_image(p[i], 5,20,width, height);

#if time_eval_filters
            gettimeofday(&t4, NULL);
            duration2 = (t4.tv_sec -t3.tv_sec)+((t4.tv_usec-t3.tv_usec)/1e6);
            fprintf(stderr,  "Blur filter done in %lf s\n", duration2);
            printf("%lf ", duration2);
            gettimeofday(&t3, NULL);
#endif

            cuda_sobel_filter_per_image(p[i], width, height);

#if time_eval_filters
            gettimeofday(&t4, NULL);
            duration2 = (t4.tv_sec -t3.tv_sec)+((t4.tv_usec-t3.tv_usec)/1e6);
            fprintf(stderr,  "Sobel filter done in %lf s\n", duration2) ;
            printf("%lf ", duration2);
#endif
        }

#if time_eval
        /* FILTER Timer stop */
        gettimeofday(&t2, NULL);

        duration = (t2.tv_sec -t1.tv_sec)+((t2.tv_usec-t1.tv_usec)/1e6);

        fprintf(stderr,  "SOBEL done in %lf s\n", duration ) ;
        printf("%lf \n", duration);
#endif
    }
}
