#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include <stdio.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

#include "basic_structure.h"

__global__ void
cuda_blur_filter_kernel(int *p, int * res, int size, int threshold, int width, int height, int* end){
    int i, j, k;
    j = threadIdx.y + blockIdx.y * blockDim.y;
    k = threadIdx.x + blockIdx.x * blockDim.x;
    i = CONV(j,k,width);
    int end1 = height/10 - size;
    int end2 = height*0.9 + size;

    if (j >= 0 && j < height-1 && k >= 0 && k < width-1)
    {
        res[i] = p[i];
    }
    __syncthreads();
    /* Apply blur on top/bottom part of image (10% & 90%) */
    if (k >= size && k < width-size){
        if ((j >= size && j < end1) || (j >= end2 && j < height-size))
        {
            int stencil_j, stencil_k ;
            int t_r = 0 ;
            for ( stencil_j = -size ; stencil_j <= size ; stencil_j++ )
            {
                for ( stencil_k = -size ; stencil_k <= size ; stencil_k++ )
                {
                    t_r += p[CONV(j+stencil_j,k+stencil_k,width)] ;
                }
            }
            res[i] = t_r / ( (2*size+1)*(2*size+1) ) ;
            /* Test the end condition on the variables that we have changed */
            float diff_r;
            diff_r = (res[i] - p[i]);
            if ( diff_r > threshold || -diff_r > threshold) {
                *end = 0;
            }
        }
        /* Copy the middle part of the image */
        else if (j >= end1 && j < end2){
            res[i] = p[i];
        }
    }
    __syncthreads();
    /* If the difference is large enough, we are going to reblur the image */
    if (j >= 1 && j < height - 1 && k >= 1 && k < width - 1)
    {
        p[i] = res[i] ;
    }
    __syncthreads();
}
__global__
void cuda_sobel_filter_kernel(int* p, int* res, int width, int height){
    int i, j, k;
    j = threadIdx.y + blockIdx.y * blockDim.y;
    k = threadIdx.x + blockIdx.x * blockDim.x;
    i = CONV(j,k,width);
    if (j >= 1 && j < height - 1 && k >= 1 && k < width-1)
    {
        int pixel_blue_no, pixel_blue_n, pixel_blue_ne;
        int pixel_blue_so, pixel_blue_s, pixel_blue_se;
        int pixel_blue_o , pixel_blue_e ;
        float deltaX_blue ;
        float deltaY_blue ;
        float val_blue;
        pixel_blue_no = p[CONV(j-1,k-1,width)] ;
        pixel_blue_n  = p[CONV(j-1,k  ,width)] ;
        pixel_blue_ne = p[CONV(j-1,k+1,width)] ;
        pixel_blue_so = p[CONV(j+1,k-1,width)] ;
        pixel_blue_s  = p[CONV(j+1,k  ,width)] ;
        pixel_blue_se = p[CONV(j+1,k+1,width)] ;
        pixel_blue_o  = p[CONV(j  ,k-1,width)] ;
        pixel_blue_e  = p[CONV(j  ,k+1,width)] ;

        deltaX_blue = -pixel_blue_no + pixel_blue_ne - 2*pixel_blue_o + 2*pixel_blue_e - pixel_blue_so + pixel_blue_se;
        deltaY_blue = pixel_blue_se + 2*pixel_blue_s + pixel_blue_so - pixel_blue_ne - 2*pixel_blue_n - pixel_blue_no;
        val_blue = sqrt(deltaX_blue * deltaX_blue + deltaY_blue * deltaY_blue)/4;
        if ( val_blue > 50 ) res[i] = 255 ;
        else res[i] = 0 ;
    }
    else if (j < height && k < height)
    {
        res[i] = p[i];
    }
    __syncthreads();
}

extern "C"
{
    void cuda_filter_per_image(int* p, int size, int threshold, int width, int height){
        hipSetDevice(0);
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, 0);
        hipEvent_t start, stop;

        int total_size = width * height * sizeof(int);
        //printf("\n %d \n", total_size);

        dim3 dimBlock(
                min(32, width), 
                min(32, height)
                );
        dim3 dimGrid(
                width / dimBlock.x + 1,
                height / dimBlock.y + 1
                );
        printf("\nSize dimBlock : %d x %d \n", dimBlock.x, dimBlock.y);
        printf("Size dimGrid : %d x %d \n", dimGrid.x, dimGrid.y);
        printf("Threads needed : %d, Threads had : %d \n", width * height, dimBlock.x*dimBlock.y*dimGrid.x * dimGrid.y);
        /* Define device variables */
        int * d_p;
        int * d_res;
        int * d_end;

        /* Allocation of memory */
        checkCudaErrors(hipEventCreate(&start));
        checkCudaErrors(hipMalloc( &d_p, total_size));
        checkCudaErrors(hipMalloc( &d_res, total_size));
        checkCudaErrors(hipMalloc( &d_end, sizeof(int)));
        checkCudaErrors(hipEventCreate(&stop));

        /* Copy array from CPU to device */
        checkCudaErrors(hipMemcpy(d_p, p, total_size, hipMemcpyHostToDevice));

        /* execute the kernel */
        int num_iter = 0;
        int end;
        do{
            end = 1;
            num_iter++;
            hipMemcpy(d_end, &end, sizeof(int), hipMemcpyHostToDevice);
            cuda_blur_filter_kernel<<<dimGrid, dimBlock>>>(d_p, d_res, size, threshold, width, height, d_end);
            hipMemcpy(&end, d_end, sizeof(int), hipMemcpyDeviceToHost);
        }while (threshold > 0 && !end);
        printf("\nBlur filtering...Done! %d \n", num_iter);

        cuda_sobel_filter_kernel<<<dimGrid, dimBlock>>>(d_p, d_res, width, height);

        /* return the result from device to CPU */
        checkCudaErrors(hipMemcpy(p, d_res, total_size, hipMemcpyDeviceToHost));
        checkCudaErrors(hipFree(d_p));
        checkCudaErrors(hipFree(d_res));
        checkCudaErrors(hipFree(d_end));
    }

    void cuda_filter( animated_gif * image){
        struct timeval t1, t2;
        double duration;

        /* FILTER Timer start */
        gettimeofday(&t1, NULL);
        fprintf(stderr, "\nUsing cuda functions\n");
        printf("%s ", "CUDA");

    // Apply cuda filter
        int i, width, height;
        int ** p;
        p = image->p;
        for(i=0; i<image->n_images; i++){
            width = image->width[i];
            height = image->height[i];
            cuda_filter_per_image(p[i], 5, 20, width, height);
        }

        /* FILTER Timer stop */
        gettimeofday(&t2, NULL);
        duration = (t2.tv_sec -t1.tv_sec)+((t2.tv_usec-t1.tv_usec)/1e6);
        fprintf(stderr,  "SOBEL done in %lf s\n", duration ) ;
        printf("%lf ", duration);
    }
}
