#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include <stdio.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

#include "basic_structure.h"

__global__ void
cuda_blur_filter_kernel(int *p, int * res, int size, int threshold, int width, int height, int* end){
    int total_size = width * height;
    int index;
    int nb_threads;
    index = blockIdx.x * blockDim.x + threadIdx.x;
    nb_threads = blockDim.x * gridDim.x;
    int i, j, k;
    int end1 = height/10 - size;
    int end2 = height*0.9 + size;

    for (i = index; i < total_size; i+= nb_threads)
    {
        j = i / width;
        k = i % width;
        if (j >= 0 && j < height-1 && k >= 0 && k < width-1)
        {
        res[i] = p[i];
        }
    }
    __syncthreads();
    for (i = index; i < total_size; i+= nb_threads)
    {
        j = i / width;
        k = i % width;
        if ((j >= size && j < end1 && k >= size && k < width-size)||
         (j >= end2 && j < height-size && k >= size && k <= width-size))
        {
            int stencil_j, stencil_k ;
            int t_r = 0 ;
            for ( stencil_j = -size ; stencil_j <= size ; stencil_j++ )
            {
                for ( stencil_k = -size ; stencil_k <= size ; stencil_k++ )
                {
                    t_r += p[CONV(j+stencil_j,k+stencil_k,width)] ;
                }
            }
            res[i] = t_r / ( (2*size+1)*(2*size+1) ) ;
        }
        else{
            res[i] = p[i];
        }
    }
    __syncthreads();
    for(i=index; i < total_size; i+= nb_threads)
    {
        j = i / width;
        k = i % width;
        if (j >= 1 && j < height - 1 && k >= 1 && k < width - 1)
        {
            float diff_r;
            diff_r = (res[i] - p[i]) ;
            if ( diff_r > threshold || -diff_r > threshold) {
                *end = 0;
            }
            p[i] = res[i] ;
        }
    }
    __syncthreads();
}
__global__
void cuda_sobel_filter_kernel(int* p, int* res, int width, int height){
    int i, j, k;
    int total_size = width * height;
    int index;
    int nb_threads;
    index = blockIdx.x * blockDim.x + threadIdx.x;
    nb_threads = blockDim.x * gridDim.x;
    for (i = index; i < total_size; i+= nb_threads)
    {
        j = i % height;
        k = i % width;
        if (j >= 1 && j < height - 1 && k >= 1 && k < width-1){
            int pixel_blue_no, pixel_blue_n, pixel_blue_ne;
            int pixel_blue_so, pixel_blue_s, pixel_blue_se;
            int pixel_blue_o , pixel_blue_e ;

            float deltaX_blue ;
            float deltaY_blue ;
            float val_blue;

            pixel_blue_no = p[CONV(j-1,k-1,width)] ;
            pixel_blue_n  = p[CONV(j-1,k  ,width)] ;
            pixel_blue_ne = p[CONV(j-1,k+1,width)] ;
            pixel_blue_so = p[CONV(j+1,k-1,width)] ;
            pixel_blue_s  = p[CONV(j+1,k  ,width)] ;
            pixel_blue_se = p[CONV(j+1,k+1,width)] ;
            pixel_blue_o  = p[CONV(j  ,k-1,width)] ;
            pixel_blue_e  = p[CONV(j  ,k+1,width)] ;

            deltaX_blue = -pixel_blue_no + pixel_blue_ne - 2*pixel_blue_o + 2*pixel_blue_e - pixel_blue_so + pixel_blue_se;             

            deltaY_blue = pixel_blue_se + 2*pixel_blue_s + pixel_blue_so - pixel_blue_ne - 2*pixel_blue_n - pixel_blue_no;

            val_blue = sqrt(deltaX_blue * deltaX_blue + deltaY_blue * deltaY_blue)/4;

            if ( val_blue > 50 ) 
            {
                res[i] = 255 ;
            } else
            {
                res[i] = 0 ;
            }
        }
        else{
            res[i] = p[i];
        }
    }
    __syncthreads();
}

extern "C"
{
    void cuda_filter_per_image(int* p, int size, int threshold, int width, int height){
        hipSetDevice(0);
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, 0);

        int total_size = width * height * sizeof(int);
        //printf("\n %d \n", total_size);

        dim3 dimBlock(deviceProp.maxThreadsPerBlock);//, deviceProp.maxThreadsDim[1]);
        dim3 dimGrid(total_size/deviceProp.maxThreadsPerBlock + 1);
        /* Define device variables */
        int * d_p;
        int * d_res;
        int * d_end;

        /* Allocation of memory */
        hipMalloc( &d_p, total_size);
        hipMalloc( &d_res, total_size);
        hipMalloc( &d_end, sizeof(int));

        /* Copy array from CPU to device */
        hipMemcpy(d_p, p, total_size, hipMemcpyHostToDevice);

        /* execute the kernel */
        int num_iter = 0;
        int end;
        do{
            end = 1;
            num_iter++;
            printf("Blur filtering... %d \n", end);
            hipMemcpy(d_end, &end, sizeof(int), hipMemcpyHostToDevice);
            cuda_blur_filter_kernel<<<dimGrid, dimBlock>>>(d_p, d_res, size, threshold, width, height, d_end);
            hipMemcpy(&end, d_end, sizeof(int), hipMemcpyDeviceToHost);
            printf("Blur filtering...Done! %d \n", end);
        }while (threshold > 0 && !end);

        //cuda_sobel_filter_kernel<<<dimGrid, dimBlock>>>(d_p, d_res, width, height);

        /* return the result from device to CPU */
        hipMemcpy(p, d_res, total_size, hipMemcpyDeviceToHost);
        hipFree(d_p);
        hipFree(d_res);
        hipFree(d_end);
    }

    void cuda_filter( animated_gif * image){
        struct timeval t1, t2;
        double duration;

        /* FILTER Timer start */
        gettimeofday(&t1, NULL);
        fprintf(stderr, "\nUsing cuda functions\n");
        printf("%s ", "CUDA");

    // Apply cuda filter
        int i, width, height;
        int ** p;
        p = image->p;
        for(i=0; i<image->n_images; i++){
            width = image->width[i];
            height = image->height[i];
            cuda_filter_per_image(p[i], 5, 20, width, height);
        }

        /* FILTER Timer stop */
        gettimeofday(&t2, NULL);
        duration = (t2.tv_sec -t1.tv_sec)+((t2.tv_usec-t1.tv_usec)/1e6);
        fprintf(stderr,  "SOBEL done in %lf s\n", duration ) ;
        printf("%lf ", duration);
    }
}
